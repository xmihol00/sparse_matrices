#include "hip/hip_runtime.h"
#include "dense.cuh"
#include "dense_priv.cuh"

using namespace std;

__global__ void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint16_t elements)
{
    uint16_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t columnIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (rowIndex >= rowsA || columnIndex >= columnsB)
    {
        return;
    }

    float *offsetOperandA = &operandA[rowIndex * elements];
    float *offsetOperandB = &operandB[columnIndex * elements];
    
    register float accumulator = 0;
    for (uint16_t i = 0; i < elements; i++)
    {
        accumulator += offsetOperandA[i] * offsetOperandB[i];
    }

    target[columnIndex * columnsB + rowIndex] = accumulator;
}

void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint64_t sizeA, uint64_t sizeB)
{
    uint8_t blockDimension = rowsA * columnsB / (UINT8_MAX + 1) > UINT16_MAX ? 32 : 16;
    float *dOperandA, *dOperandB, *dTarget;
    uint64_t sizeTraget = rowsA * columnsB * sizeof(float);
    
    hipMalloc(reinterpret_cast<void **>(&dOperandA), sizeA);
    hipMalloc(reinterpret_cast<void **>(&dOperandB), sizeB);
    hipMalloc(reinterpret_cast<void **>(&dTarget), sizeTraget);
    
    hipMemcpy(dOperandA, operandA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dOperandB, operandB, sizeB, hipMemcpyHostToDevice);

    dim3 blockSize(blockDimension, blockDimension, 1);
    dim3 gridSize((rowsA + blockDimension - 1) / blockDimension, (columnsB + blockDimension - 1) / blockDimension, 1);
    dotRowsColumns<<<gridSize, blockSize>>>(dOperandA, dOperandB, dTarget, rowsA, columnsB, sizeA / (rowsA * sizeof(float)));
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cerr << hipGetErrorString(error) << endl;
        exit(1);
    }
    
    hipMemcpy(target, dTarget, sizeTraget, hipMemcpyDeviceToHost);
    
    hipFree(dOperandA);
    hipFree(dOperandB);
    hipFree(dTarget);
}


void dotCuBLAS(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint16_t columnsARowsB, 
                uint16_t lda, uint16_t ldb, uint16_t ldc)
{
    float *dOperandA, *dOperandB, *dTarget;
    uint32_t sizeA = rowsA * columnsARowsB * sizeof(float);
    uint32_t sizeB = columnsB * columnsARowsB * sizeof(float);
    uint32_t sizeTarget = rowsA * columnsB * sizeof(float);
    hipMalloc(reinterpret_cast<void **>(&dOperandA), sizeA);
    hipMalloc(reinterpret_cast<void **>(&dOperandB), sizeB);
    hipMalloc(reinterpret_cast<void **>(&dTarget), sizeTarget);
    
    hipMemcpy(dOperandA, operandA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dOperandB, operandB, sizeB, hipMemcpyHostToDevice);

    const float alpha = 1;
    const float beta = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rowsA, columnsB, columnsARowsB, 
                                        &alpha, dOperandA, lda, dOperandB, ldb, &beta, dTarget, ldc);
    hipError_t error = hipDeviceSynchronize();
    hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS || error != hipSuccess)
    {
        cerr << "cuBLAS matrix multiply failed: " << hipGetErrorString(error) << endl;
        exit(1);
    }

    hipMemcpy(target, dTarget, sizeTarget, hipMemcpyDeviceToHost);
    
    hipFree(dOperandA);
    hipFree(dOperandB);
    hipFree(dTarget);
}
