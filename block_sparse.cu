#include "hip/hip_runtime.h"
#include "block_sparse_cuda.h"
#include "block_sparse_cuda_priv.h"

using namespace Matrix;
using namespace std;

__global__ void dotRowsColumns(uint16_t *blockIndicesOperandA, uint8_t *entryIndicesOperandA, float *dataOperandA, float *operandB, float *target, 
                               uint16_t rowsA, uint16_t columnsB, uint16_t blocksPerRow, uint8_t entriesPerBlock, uint16_t elements)
{
    uint16_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t columnIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (rowIndex >= rowsA || columnIndex >= columnsB)
    {
        return;
    }

    uint16_t *offsetBlockIndicesOperandA = &blockIndicesOperandA[rowIndex * blocksPerRow];
    uint8_t *offsetEntryIndicesOperandA = &entryIndicesOperandA[rowIndex * blocksPerRow * (entriesPerBlock - 1)];
    float *offsetOperandA = &dataOperandA[rowIndex * blocksPerRow * entriesPerBlock];
    
    register float accumulator = 0;
    for (uint16_t i = 0; i < blocksPerRow; i++)
    {
        float *offsetOperandB = &operandB[columnIndex * elements + offsetBlockIndicesOperandA[i]];
        for (uint8_t j = 0, k = 0; j < entriesPerBlock; k = offsetEntryIndicesOperandA[j++])
        {
             accumulator += offsetOperandA[j] * offsetOperandB[k];
        }
        offsetEntryIndicesOperandA = &offsetEntryIndicesOperandA[entriesPerBlock - 1];
        offsetOperandA = &offsetOperandA[entriesPerBlock];
    }

    target[columnIndex * columnsB + rowIndex] = accumulator;
}

void dotRowsColumns(matrix_ptrs_t operandA, float *operandB, float *target, uint32_t offsetOfEntryIndices, uint64_t offsetOfData,
                    uint16_t blocksPerRow, uint8_t entriesPerBlock, uint16_t rowsA, uint16_t columnsB, uint64_t sizeA, uint64_t sizeB)
{
    uint8_t blockDimension = rowsA * columnsB / (UINT8_MAX + 1) > UINT16_MAX ? 32 : 16;
    float *dOperandB, *dTarget, *dDataOperandA;
    matrix_ptrs_t dOperandA, tmp;
    uint8_t *dEntryIndicesOperandA;
    uint64_t sizeTraget = rowsA * columnsB * sizeof(float);
    
    hipMalloc(reinterpret_cast<void **>(&dOperandA.uint8s), sizeA);
    hipMalloc(reinterpret_cast<void **>(&dOperandB), sizeB);
    hipMalloc(reinterpret_cast<void **>(&dTarget), sizeTraget);
    
    hipMemcpy(dOperandA.uint8s, operandA.uint8s, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dOperandB, operandB, sizeB, hipMemcpyHostToDevice);
    tmp.uint16s = &dOperandA.uint16s[offsetOfEntryIndices];
    dEntryIndicesOperandA = tmp.uint8s;
    tmp.uint8s = &dOperandA.uint8s[offsetOfData];
    dDataOperandA = tmp.floats;

    dim3 blockSize(blockDimension, blockDimension, 1);
    dim3 gridSize((rowsA + blockDimension - 1) / blockDimension, (columnsB + blockDimension - 1) / blockDimension, 1);
    dotRowsColumns<<<gridSize, blockSize>>>(dOperandA.uint16s, dEntryIndicesOperandA, dDataOperandA, dOperandB, dTarget, 
                                            rowsA, columnsB, blocksPerRow, entriesPerBlock, sizeB / (columnsB * sizeof(float)));
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cerr << hipGetErrorString(error) << endl;
        exit(1);
    }
    
    hipMemcpy(target, dTarget, sizeTraget, hipMemcpyDeviceToHost);
    
    hipFree(dOperandA.uint8s);
    hipFree(dOperandB);
    hipFree(dTarget);
}
