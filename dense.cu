#include "hip/hip_runtime.h"
#include "dense_cuda.h"
#include "dense_cuda_priv.h"

__global__ void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint16_t elements)
{
    uint16_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t columnIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (rowIndex >= rowsA || columnIndex >= columnsB)
    {
        return;
    }

    float *offsetOperandA = &operandA[rowIndex * elements];
    float *offsetOperandB = &operandB[columnIndex * elements];
    
    float accumulator = 0;
    for (uint16_t i = 0; i < elements; i++)
    {
        accumulator += offsetOperandA[i] * offsetOperandB[i];
    }

    target[columnIndex * columnsB + rowIndex] = accumulator;
}

void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint64_t sizeA, uint64_t sizeB)
{
    const uint8_t BLOCK_DIMENSION = 16;
    float *dOperandA, *dOperandB, *dTarget;
    uint32_t sizeTraget = rowsA * columnsB * sizeof(float);
    
    hipMalloc((void **)&dOperandA, sizeA);
    hipMalloc((void **)&dOperandB, sizeB);
    hipMalloc((void **)&dTarget, sizeTraget);
    
    hipMemcpy(dOperandA, operandA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dOperandB, operandB, sizeB, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_DIMENSION, BLOCK_DIMENSION, 1);
    dim3 gridSize((rowsA + BLOCK_DIMENSION - 1) / BLOCK_DIMENSION, (columnsB + BLOCK_DIMENSION - 1) / BLOCK_DIMENSION, 1);
    dotRowsColumns<<<blockSize, gridSize>>>(dOperandA, dOperandB, dTarget, rowsA, columnsB, sizeA / (rowsA * sizeof(float)));
    hipDeviceSynchronize();
    
    hipMemcpy(target, dTarget, sizeTraget, hipMemcpyDeviceToHost);
    
    hipFree(dOperandA);
    hipFree(dOperandB);
    hipFree(dTarget);
}
