#include "hip/hip_runtime.h"
#include "dense_cuda.h"
#include "dense_cuda_priv.h"

using namespace std;

__global__ void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint16_t elements)
{
    uint16_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint16_t columnIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (rowIndex >= rowsA || columnIndex >= columnsB)
    {
        return;
    }

    float *offsetOperandA = &operandA[rowIndex * elements];
    float *offsetOperandB = &operandB[columnIndex * elements];
    
    register float accumulator = 0;
    for (uint16_t i = 0; i < elements; i++)
    {
        accumulator += offsetOperandA[i] * offsetOperandB[i];
    }

    target[columnIndex * columnsB + rowIndex] = accumulator;
}

void dotRowsColumns(float *operandA, float *operandB, float *target, uint16_t rowsA, uint16_t columnsB, uint64_t sizeA, uint64_t sizeB)
{
    uint8_t blockDimension = rowsA * columnsB / (UINT8_MAX + 1) > UINT16_MAX ? 32 : 16;
    float *dOperandA, *dOperandB, *dTarget;
    uint64_t sizeTraget = rowsA * columnsB * sizeof(float);
    
    hipMalloc(reinterpret_cast<void **>(&dOperandA), sizeA);
    hipMalloc(reinterpret_cast<void **>(&dOperandB), sizeB);
    hipMalloc(reinterpret_cast<void **>(&dTarget), sizeTraget);
    
    hipMemcpy(dOperandA, operandA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dOperandB, operandB, sizeB, hipMemcpyHostToDevice);

    dim3 blockSize(blockDimension, blockDimension, 1);
    dim3 gridSize((rowsA + blockDimension - 1) / blockDimension, (columnsB + blockDimension - 1) / blockDimension, 1);
    dotRowsColumns<<<gridSize, blockSize>>>(dOperandA, dOperandB, dTarget, rowsA, columnsB, sizeA / (rowsA * sizeof(float)));
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        cerr << hipGetErrorString(error) << endl;
        exit(1);
    }
    
    hipMemcpy(target, dTarget, sizeTraget, hipMemcpyDeviceToHost);
    
    hipFree(dOperandA);
    hipFree(dOperandB);
    hipFree(dTarget);
}
